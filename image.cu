#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

__global__ void process(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    uchar3 full = make_uchar3(255, 255, 255);


    if (dst_x < cols && dst_y < rows)
    {
        uchar3 val = src(dst_y, dst_x);

        dst(dst_y, dst_x).x = full.x - val.x;
        dst(dst_y, dst_x).y = full.y - val.y;
        dst(dst_y, dst_x).z = full.z - val.z;
    }
}

__global__ void gaussianConv(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, cv::cuda::PtrStep<float> d_kernelGaussConv, int kernelSize, int sigma)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    const float k = (kernelSize - 1.0) / 2.0;

    if (dst_x < cols - k && dst_y < rows - k &&
        dst_x > k && dst_y > k)
    {
        float tmp[3] = { 0.0, 0.0, 0.0, };

        for (int u = dst_x - k; u <= dst_x + k; u++)
        {
            for (int v = dst_y - k; v <= dst_y + k; v++)
            {
                tmp[0] += (float)src(v, u).x * d_kernelGaussConv(v - dst_y + k, u - dst_x + k);
            
                tmp[1] += (float)src(v, u).y * d_kernelGaussConv(v - dst_y + k, u - dst_x + k);

                tmp[2] += (float)src(v, u).z * d_kernelGaussConv(v - dst_y + k, u - dst_x + k);
            }
        }
        dst(dst_y, dst_x).x = (unsigned char)(tmp[0]);
        dst(dst_y, dst_x).y = (unsigned char)(tmp[1]);
        dst(dst_y, dst_x).z = (unsigned char)(tmp[2]);
        /*
        dst(dst_y, dst_x).x = src(dst_y, dst_x).x;
        dst(dst_y, dst_x).y = src(dst_y, dst_x).y;
        dst(dst_y, dst_x).z = src(dst_y, dst_x).z;*/
    }

}

__global__ void laplacianFilter(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols)
{
    
    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (dst_x < cols - 1 && dst_y < rows - 1 &&
        dst_x > 1 && dst_y > 1)
    {
        
        // Sum of pixel values 
        float sum[3] = { 0.0, 0.0, 0.0 };
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                sum[0] += (float)(src(dst_y + i, dst_x + j).x * -1.0);
                sum[1] += (float)(src(dst_y + i, dst_x + j).y * -1.0);
                sum[2] += (float)(src(dst_y + i, dst_x + j).z * -1.0);
            }
        }
        dst(dst_y, dst_x).x = sum[0] + (src(dst_y, dst_x).x * 9.0);
        dst(dst_y, dst_x).y = sum[1] + (src(dst_y, dst_x).y * 9.0);
        dst(dst_y, dst_x).z = sum[2] + (src(dst_y, dst_x).z * 9.0);

    }
}


__global__ void imageCombination(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, const cv::cuda::PtrStep<uchar3> src2, int rows, int cols, int imageComb, float offSet, float scaleFactor)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dst_x < cols && dst_y < rows)
    {
        if (imageComb == 0) {
            dst(dst_y, dst_x).x = src(dst_y, dst_x).x + src2(dst_y, dst_x).x;
            dst(dst_y, dst_x).y = src(dst_y, dst_x).y + src2(dst_y, dst_x).y;
            dst(dst_y, dst_x).z = src(dst_y, dst_x).z + src2(dst_y, dst_x).z;
        }
        else if (imageComb == 1) {
            dst(dst_y, dst_x).x = src(dst_y, dst_x).x - src2(dst_y, dst_x).x;
            dst(dst_y, dst_x).y = src(dst_y, dst_x).y - src2(dst_y, dst_x).y;
            dst(dst_y, dst_x).z = src(dst_y, dst_x).z - src2(dst_y, dst_x).z;
        }
        else if (imageComb == 2) {
            dst(dst_y, dst_x).x = src(dst_y, dst_x).x * src2(dst_y, dst_x).x;
            dst(dst_y, dst_x).y = src(dst_y, dst_x).y * src2(dst_y, dst_x).y;
            dst(dst_y, dst_x).z = src(dst_y, dst_x).z * src2(dst_y, dst_x).z;
        }
        else if (imageComb == 3) {
            dst(dst_y, dst_x).x = src2(dst_y, dst_x).x == 0 ? src(dst_y, dst_x).x : src(dst_y, dst_x).x / src2(dst_y, dst_x).x;
            dst(dst_y, dst_x).y = src2(dst_y, dst_x).y == 0 ? src(dst_y, dst_x).y : src(dst_y, dst_x).y / src2(dst_y, dst_x).y;
            dst(dst_y, dst_x).z = src2(dst_y, dst_x).z == 0 ? src(dst_y, dst_x).z : src(dst_y, dst_x).z / src2(dst_y, dst_x).z;
        }

        dst(dst_y, dst_x).x *= scaleFactor;
        dst(dst_y, dst_x).y *= scaleFactor;
        dst(dst_y, dst_x).z *= scaleFactor;

        dst(dst_y, dst_x).x += offSet;
        dst(dst_y, dst_x).y += offSet;
        dst(dst_y, dst_x).z += offSet;
    }
}

int divUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY)
{
    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    process << <grid, block >> > (src, dst, dst.rows, dst.cols);

}

void gaussianConvCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY, cv::cuda::GpuMat& d_kernelGaussConv, int kernelSize, int sigma)
{

    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    gaussianConv << <grid, block >> > (src, dst, dst.rows, dst.cols, d_kernelGaussConv, kernelSize, sigma);

}

void laplacianCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY)
{

    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    laplacianFilter << <grid, block >> > (src, dst, dst.rows, dst.cols);

}

void imageCombCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, cv::cuda::GpuMat& src2, int dimX, int dimY, int imageComb, float offSet, float scaleFactor)
{

    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    imageCombination << <grid, block >> > (src, dst, src2, dst.rows, dst.cols, imageComb, offSet, scaleFactor);

}

