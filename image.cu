#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

__global__ void process(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    uchar3 full = make_uchar3(255, 255, 255);


    if (dst_x < cols && dst_y < rows)
    {
        uchar3 val = src(dst_y, dst_x);

        dst(dst_y, dst_x).x = full.x - val.x;
        dst(dst_y, dst_x).y = full.y - val.y;
        dst(dst_y, dst_x).z = full.z - val.z;
    }
}

__global__ void gaussian(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    uchar3 full = make_uchar3(255, 255, 255);


    if (dst_x < cols && dst_y < rows)
    {
        uchar3 val = src(dst_y, dst_x);

        dst(dst_y, dst_x).x = full.x - val.x;
        dst(dst_y, dst_x).y = full.y - val.y;
        dst(dst_y, dst_x).z = full.z - val.z;
    }
}

int divUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY)
{
    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    process << <grid, block >> > (src, dst, dst.rows, dst.cols);

}

void gaussianCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY)
{

    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    gaussian << <grid, block >> > (src, dst, dst.rows, dst.cols);

}

