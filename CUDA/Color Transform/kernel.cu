#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

__device__ float3 bgr2xyz(uchar3 src) {

    float scr_r = src.z / 255.0;
    float scr_g = src.y / 255.0;
    float scr_b = src.x / 255.0;

    float tmp[3];
    tmp[0] = 100.0 * ((scr_r > .04045) ? pow((scr_r + .055) / 1.055, 2.4) : scr_r / 12.92);
    tmp[1] = 100.0 * ((scr_g > .04045) ? pow((scr_g + .055) / 1.055, 2.4) : scr_g / 12.92);
    tmp[2] = 100.0 * ((scr_b > .04045) ? pow((scr_b + .055) / 1.055, 2.4) : scr_b / 12.92);

    float3 xyz;
    xyz.x = .4124 * tmp[0] + .3576 * tmp[1] + .1805 * tmp[2];
    xyz.y = .2126 * tmp[0] + .7152 * tmp[1] + .0722 * tmp[2];
    xyz.z = .0193 * tmp[0] + .1192 * tmp[1] + .9505 * tmp[2];

    return xyz;
}

__device__ float3 xyz2lab(float3 src, float angle) {

    float scr_z = src.z / 108.883;
    float scr_y = src.y / 100.;
    float scr_x = src.x / 95.047;

    float PI = 3.14159265358979323846;

    float v[3];
    v[0] = (scr_x > .008856) ? pow(scr_x, 1. / 3.) : (7.787 * scr_x) + (16. / 116.);
    v[1] = (scr_y > .008856) ? pow(scr_y, 1. / 3.) : (7.787 * scr_y) + (16. / 116.);
    v[2] = (scr_z > .008856) ? pow(scr_z, 1. / 3.) : (7.787 * scr_z) + (16. / 116.);

    float3 lab;
    lab.x = (116. * v[1]) - 16.;
    lab.y = 500. * (v[0] - v[1]);
    lab.z = 200. * (v[1] - v[2]);

    float C = sqrt(pow(lab.y, 2) + pow(lab.z, 2));
    float h = atan2(lab.z, lab.y);
    h += (angle * PI) / 180.0;
    lab.y = cos(h) * C;
    lab.z = sin(h) * C;

    return lab;
}

__device__ float3 bgr2lab(uchar3 c, float angle) {
    return xyz2lab(bgr2xyz(c), angle);
}

__device__ float3 lab2xyz(float3 src) {

    float fy = (src.x + 16.0) / 116.0;
    float fx = src.y / 500.0 + fy;
    float fz = fy - src.z / 200.0;

    float3 lab;
    lab.x = 95.047 * ((fx > 0.206897) ? fx * fx * fx : (fx - 16.0 / 116.0) / 7.787);
    lab.y = 100.000 * ((fy > 0.206897) ? fy * fy * fy : (fy - 16.0 / 116.0) / 7.787);
    lab.z = 108.883 * ((fz > 0.206897) ? fz * fz * fz : (fz - 16.0 / 116.0) / 7.787);

    return lab;
}

__device__ float3 xyz2bgr(float3 src) {

    src.x /= 100.0;
    src.y /= 100.0;
    src.z /= 100.0;


    float tmp[3];

    tmp[0] = 3.2406 * src.x - 1.5372 * src.y - 0.4986 * src.z;
    tmp[1] = -0.9689 * src.x + 1.8758 * src.y + 0.0415 * src.z;
    tmp[2] = 0.0557 * src.x - 0.2040 * src.y + 1.0570 * src.z;

    float3 bgr;
    bgr.z = 255.0 * ((tmp[0] > 0.0031308) ? ((1.055 * pow(tmp[0], (1.0 / 2.4))) - 0.055) : 12.92 * (tmp[0]));
    bgr.y = 255.0 * ((tmp[1] > 0.0031308) ? ((1.055 * pow(tmp[1], (1.0 / 2.4))) - 0.055) : 12.92 * (tmp[1]));
    bgr.x = 255.0 * ((tmp[2] > 0.0031308) ? ((1.055 * pow(tmp[2], (1.0 / 2.4))) - 0.055) : 12.92 * (tmp[2]));

    return bgr;
}

__device__ float3 lab2bgr(float3 src) {
    return xyz2bgr(lab2xyz(src));
}

__global__ void hueShift(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, float angle)
{

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dst_x < cols && dst_y < rows)
    {
        float3 bgr;
        bgr.x = lab2bgr(bgr2lab(src(dst_y, dst_x), angle)).x;
        bgr.y = lab2bgr(bgr2lab(src(dst_y, dst_x), angle)).y;
        bgr.z = lab2bgr(bgr2lab(src(dst_y, dst_x), angle)).z;

        dst(dst_y, dst_x).x = (unsigned char)(bgr.x < 0 ? 0 : (bgr.x > 255 ? 255 : bgr.x));
        dst(dst_y, dst_x).y = (unsigned char)(bgr.y < 0 ? 0 : (bgr.y > 255 ? 255 : bgr.y));
        dst(dst_y, dst_x).z = (unsigned char)(bgr.z < 0 ? 0 : (bgr.z > 255 ? 255 : bgr.z));

    }

}

int divUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void colorTransfCUDA(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int dimX, int dimY, float angle)
{

    const dim3 block(dimX, dimY);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    hueShift << <grid, block >> > (src, dst, dst.rows, dst.cols, angle);

}
